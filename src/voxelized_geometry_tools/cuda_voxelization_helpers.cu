#include "hip/hip_runtime.h"
#include <voxelized_geometry_tools/cuda_voxelization_helpers.h>

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace voxelized_geometry_tools
{
namespace pointcloud_voxelization
{
namespace cuda_helpers
{
bool IsAvailable() { return true; }

#define CudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

__global__
void RaycastPoint(
    const float* const device_points_ptr, const int32_t num_points,
    const float* const device_pointcloud_origin_transform_ptr,
    const float* const device_tracking_grid_inverse_origin_transform_ptr,
    const float inverse_step_size, const float inverse_cell_size,
    const int32_t num_x_cells, const int32_t num_y_cells,
    const int32_t num_z_cells, const int32_t stride1, const int32_t stride2,
    int32_t* const device_tracking_grid_ptr)
{
  const int32_t point_index = blockIdx.x * blockDim.x + threadIdx.x;
  if (point_index < num_points)
  {
    const float ox = device_pointcloud_origin_transform_ptr[12];
    const float oy = device_pointcloud_origin_transform_ptr[13];
    const float oz = device_pointcloud_origin_transform_ptr[14];
    const float px = device_points_ptr[(point_index * 3) + 0];
    const float py = device_points_ptr[(point_index * 3) + 1];
    const float pz = device_points_ptr[(point_index * 3) + 2];
    const float wx = device_pointcloud_origin_transform_ptr[0] * px
                     + device_pointcloud_origin_transform_ptr[4] * py
                     + device_pointcloud_origin_transform_ptr[8] * pz
                     + device_pointcloud_origin_transform_ptr[12];
    const float wy = device_pointcloud_origin_transform_ptr[1] * px
                     + device_pointcloud_origin_transform_ptr[5] * py
                     + device_pointcloud_origin_transform_ptr[9] * pz
                     + device_pointcloud_origin_transform_ptr[13];
    const float wz = device_pointcloud_origin_transform_ptr[2] * px
                     + device_pointcloud_origin_transform_ptr[6] * py
                     + device_pointcloud_origin_transform_ptr[10] * pz
                     + device_pointcloud_origin_transform_ptr[14];
    const float rx = wx - ox;
    const float ry = wy - oy;
    const float rz = wz - oz;
    const float current_ray_length = sqrtf((rx * rx) + (ry * ry) + (rz * rz));
    const float num_steps =
        floor(current_ray_length * inverse_step_size);
    int32_t previous_x_cell = -1;
    int32_t previous_y_cell = -1;
    int32_t previous_z_cell = -1;
    for (float step = 0.0; step < num_steps; step += 1.0)
    {
      bool in_grid = false;
      const float elapsed_ratio = step / num_steps;
      const float cx = (rx * elapsed_ratio) + ox;
      const float cy = (ry * elapsed_ratio) + oy;
      const float cz = (rz * elapsed_ratio) + oz;
      const float gx =
          device_tracking_grid_inverse_origin_transform_ptr[0] * cx
          + device_tracking_grid_inverse_origin_transform_ptr[4] * cy
          + device_tracking_grid_inverse_origin_transform_ptr[8] * cz
          + device_tracking_grid_inverse_origin_transform_ptr[12];
      const float gy =
          device_tracking_grid_inverse_origin_transform_ptr[1] * cx
          + device_tracking_grid_inverse_origin_transform_ptr[5] * cy
          + device_tracking_grid_inverse_origin_transform_ptr[9] * cz
          + device_tracking_grid_inverse_origin_transform_ptr[13];
      const float gz =
          device_tracking_grid_inverse_origin_transform_ptr[2] * cx
          + device_tracking_grid_inverse_origin_transform_ptr[6] * cy
          + device_tracking_grid_inverse_origin_transform_ptr[10] * cz
          + device_tracking_grid_inverse_origin_transform_ptr[14];
      const int32_t x_cell = static_cast<int32_t>(gx * inverse_cell_size);
      const int32_t y_cell = static_cast<int32_t>(gy * inverse_cell_size);
      const int32_t z_cell = static_cast<int32_t>(gz * inverse_cell_size);
      if (x_cell != previous_x_cell || y_cell != previous_y_cell
          || z_cell != previous_z_cell)
      {
        if (x_cell >= 0 && x_cell < num_x_cells && y_cell >= 0
            && y_cell < num_y_cells && z_cell >= 0 && z_cell < num_z_cells)
        {
          in_grid = true;
          const int32_t cell_index =
              (x_cell * stride1) + (y_cell * stride2) + z_cell;
          // Increase free count
          atomicAdd(&(device_tracking_grid_ptr[cell_index * 2]), 1);
        }
        else if (in_grid)
        {
          // We've left the grid and there's no reason to keep going.
          break;
        }
      }
      previous_x_cell = x_cell;
      previous_y_cell = y_cell;
      previous_z_cell = z_cell;
    }
    // Set the point itself as filled
    const float gx =
        device_tracking_grid_inverse_origin_transform_ptr[0] * wx
        + device_tracking_grid_inverse_origin_transform_ptr[4] * wy
        + device_tracking_grid_inverse_origin_transform_ptr[8] * wz
        + device_tracking_grid_inverse_origin_transform_ptr[12];
    const float gy =
        device_tracking_grid_inverse_origin_transform_ptr[1] * wx
        + device_tracking_grid_inverse_origin_transform_ptr[5] * wy
        + device_tracking_grid_inverse_origin_transform_ptr[9] * wz
        + device_tracking_grid_inverse_origin_transform_ptr[13];
    const float gz =
        device_tracking_grid_inverse_origin_transform_ptr[2] * wx
        + device_tracking_grid_inverse_origin_transform_ptr[6] * wy
        + device_tracking_grid_inverse_origin_transform_ptr[10] * wz
        + device_tracking_grid_inverse_origin_transform_ptr[14];
    const int32_t x_cell = static_cast<int32_t>(gx * inverse_cell_size);
    const int32_t y_cell = static_cast<int32_t>(gy * inverse_cell_size);
    const int32_t z_cell = static_cast<int32_t>(gz * inverse_cell_size);
    if (x_cell >= 0 && x_cell < num_x_cells && y_cell >= 0
        && y_cell < num_y_cells && z_cell >= 0 && z_cell < num_z_cells)
    {
      const int32_t cell_index =
          (x_cell * stride1) + (y_cell * stride2) + z_cell;
      // Increase filled count
      atomicAdd(&(device_tracking_grid_ptr[(cell_index * 2) + 1]), 1);
    }
  }
}

__global__
void FilterGrids(
    const int64_t num_cells, const int32_t num_device_tracking_grids,
    int32_t* const* device_tracking_grid_ptrs,
    float* const device_filter_grid_ptr, const float percent_seen_free,
    const int32_t outlier_points_threshold, const int32_t num_cameras_seen_free)
{
  const int32_t voxel_index = blockIdx.x * blockDim.x + threadIdx.x;
  if (voxel_index < num_cells)
  {
    const float current_occupancy = device_filter_grid_ptr[voxel_index * 2];
    // Filled cells stay filled, we don't work with them.
    // We only change cells that are unknown or empty.
    if (current_occupancy <= 0.5)
    {
      int32_t cameras_seen_filled = 0;
      int32_t cameras_seen_free = 0;
      for (int32_t idx = 0; idx < num_device_tracking_grids; idx++)
      {
        int32_t* const device_tracking_grid_ptr =
            device_tracking_grid_ptrs[idx];
        const int32_t free_count = device_tracking_grid_ptr[voxel_index * 2];
        const int32_t filled_count =
            device_tracking_grid_ptr[(voxel_index * 2) + 1];
        const int32_t filtered_filled_count =
            (filled_count >= outlier_points_threshold) ? filled_count : 0;
        if (free_count > 0 && filtered_filled_count > 0)
        {
          const float current_percent_seen_free =
              static_cast<float>(free_count)
              / static_cast<float>(free_count + filtered_filled_count);
          if (current_percent_seen_free >= percent_seen_free)
          {
            cameras_seen_free += 1;
          }
          else
          {
            cameras_seen_filled += 1;
          }
        }
        else if (free_count > 0)
        {
          cameras_seen_free += 1;
        }
        else if (filtered_filled_count > 0)
        {
          cameras_seen_filled += 1;
        }
      }
      if (cameras_seen_filled > 0)
      {
        // If any camera saw something here, it is filled.
        device_filter_grid_ptr[voxel_index * 2] = 1.0;
      }
      else if (cameras_seen_free >= num_cameras_seen_free)
      {
        // Did enough cameras see this empty?
        device_filter_grid_ptr[voxel_index * 2] = 0.0;
      }
      else
      {
        // Otherwise, it is unknown.
        device_filter_grid_ptr[voxel_index * 2] = 0.5;
      }
    }
  }
}

float* PreparePointCloud(const int32_t num_points, const float* points)
{
  const size_t points_size = sizeof(float) * num_points * 3;
  float* device_points_ptr = nullptr;
  hipMalloc(&device_points_ptr, points_size);
  CudaCheckErrors("Failed to allocate device points");
  hipMemcpy(device_points_ptr, points, points_size,
             hipMemcpyHostToDevice);
  CudaCheckErrors("Failed to memcpy the points to the device");
  return device_points_ptr;
}

int32_t* PrepareTrackingGrid(const int64_t num_cells)
{
  const size_t tracking_grid_size = sizeof(int32_t) * num_cells * 2;
  int32_t* device_tracking_grid_ptr = nullptr;
  hipMalloc(&device_tracking_grid_ptr, tracking_grid_size);
  CudaCheckErrors("Failed to allocate device tracking grid");
  hipMemset(device_tracking_grid_ptr, 0, tracking_grid_size);
  CudaCheckErrors("Failed to zero device tracking grid");
  return device_tracking_grid_ptr;
}

void RaycastPoints(
    const float* const device_points_ptr, const int32_t num_points,
    const float* const pointcloud_origin_transform,
    const float* const inverse_grid_origin_transform,
    const float inverse_step_size, const float inverse_cell_size,
    const int32_t num_x_cells, const int32_t num_y_cells,
    const int32_t num_z_cells, int32_t* const device_tracking_grid_ptr)
{
  // Copy pointcloud origin transform
  const size_t transform_size = sizeof(float) * 16;
  float* device_pointcloud_origin_transform_ptr = nullptr;
  hipMalloc(&device_pointcloud_origin_transform_ptr, transform_size);
  CudaCheckErrors("Failed to allocate device pointcloud origin transform");
  hipMemcpy(
      device_pointcloud_origin_transform_ptr, pointcloud_origin_transform,
      transform_size, hipMemcpyHostToDevice);
  CudaCheckErrors("Failed to memcpy the pointcloud origin transform");
  // Copy grid inverse origin transform
  float* device_tracking_grid_inverse_origin_transform_ptr = nullptr;
  hipMalloc(
      &device_tracking_grid_inverse_origin_transform_ptr, transform_size);
  CudaCheckErrors("Failed to allocate device grid inverse origin transform");
  hipMemcpy(
      device_tracking_grid_inverse_origin_transform_ptr,
      inverse_grid_origin_transform, transform_size, hipMemcpyHostToDevice);
  CudaCheckErrors("Failed to memcpy the grid inverse origin transform");
  // Prepare for raycasting
  const int32_t stride1 = num_y_cells * num_z_cells;
  const int32_t stride2 = num_z_cells;
  // Call the CUDA kernel
  const int32_t num_threads = 256;
  const int32_t num_blocks = (num_points + (num_threads - 1)) / num_threads;
  RaycastPoint<<<num_blocks, num_threads>>>(
      device_points_ptr, num_points, device_pointcloud_origin_transform_ptr,
      device_tracking_grid_inverse_origin_transform_ptr,
      inverse_step_size, inverse_cell_size, num_x_cells, num_y_cells,
      num_z_cells, stride1, stride2, device_tracking_grid_ptr);
  // Free the device memory
  hipFree(device_pointcloud_origin_transform_ptr);
  CudaCheckErrors("Failed to free device pointcloud origin transform");
  hipFree(device_tracking_grid_inverse_origin_transform_ptr);
  CudaCheckErrors(
      "Failed to free device tracking grid inverse origin tranform");
}

float* PrepareFilterGrid(const int64_t num_cells, const void* host_data_ptr)
{
  const size_t filter_grid_size = sizeof(float) * num_cells * 2;
  float* device_filter_grid_ptr = nullptr;
  hipMalloc(&device_filter_grid_ptr, filter_grid_size);
  CudaCheckErrors("Failed to allocate device filter grid");
  hipMemcpy(device_filter_grid_ptr, host_data_ptr, filter_grid_size,
             hipMemcpyHostToDevice);
  CudaCheckErrors("Failed to memcpy the static environment to the device");
  return device_filter_grid_ptr;
}

void FilterTrackingGrids(
    const int64_t num_cells, const int32_t num_device_tracking_grids,
    int32_t* const* device_tracking_grid_ptrs,
    float* const device_filter_grid_ptr, const float percent_seen_free,
    const int32_t outlier_points_threshold, const int32_t num_cameras_seen_free)
{
  const size_t device_tracking_grid_ptrs_size =
      sizeof(int32_t*) * num_device_tracking_grids;
  int32_t** device_tracking_grid_ptrs_ptr = nullptr;
  hipMalloc(&device_tracking_grid_ptrs_ptr, device_tracking_grid_ptrs_size);
  CudaCheckErrors("Failed to allocate device tracking grid ptr storage");
  hipMemcpy(device_tracking_grid_ptrs_ptr, device_tracking_grid_ptrs,
             device_tracking_grid_ptrs_size, hipMemcpyHostToDevice);
  CudaCheckErrors("Failed to memcpy the device tracking grid ptrs to device");
  // Call the CUDA kernel
  const int32_t num_threads = 256;
  const int32_t num_blocks = (num_cells + (num_threads - 1)) / num_threads;
  FilterGrids<<<num_blocks, num_threads>>>(
      num_cells, num_device_tracking_grids, device_tracking_grid_ptrs_ptr,
      device_filter_grid_ptr, percent_seen_free, outlier_points_threshold,
      num_cameras_seen_free);
  // Free the device memory
  hipFree(device_tracking_grid_ptrs_ptr);
  CudaCheckErrors("Failed to free device tracking grid ptr storage");
}

void RetrieveTrackingGrid(
    const int64_t num_cells, const int32_t* const device_tracking_grid_ptr,
    void* host_data_ptr)
{
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  const size_t tracking_grid_size = sizeof(int32_t) * num_cells * 2;
  hipMemcpy(host_data_ptr, device_tracking_grid_ptr, tracking_grid_size,
             hipMemcpyDeviceToHost);
  CudaCheckErrors("Failed to memcpy the tracking grid back to the host");
}

void RetrieveFilteredGrid(
    const int64_t num_cells, const float* const device_filter_grid_ptr,
    void* host_data_ptr)
{
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  const size_t filter_grid_size = sizeof(float) * num_cells * 2;
  hipMemcpy(host_data_ptr, device_filter_grid_ptr, filter_grid_size,
             hipMemcpyDeviceToHost);
  CudaCheckErrors("Failed to memcpy the filter grid back to the host");
}

void CleanupDeviceMemory(
    const int32_t num_device_pointclouds, float* const* device_pointcloud_ptrs,
    const int32_t num_device_tracking_grids,
    int32_t* const* device_tracking_grid_ptrs, float* device_filter_grid_ptr)
{
  for (int32_t idx = 0; idx < num_device_pointclouds; idx++)
  {
    auto device_pointcloud_ptr = device_pointcloud_ptrs[idx];
    hipFree(device_pointcloud_ptr);
    CudaCheckErrors("Failed to free device points");
  }
  for (int32_t idx = 0; idx < num_device_tracking_grids; idx++)
  {
    auto device_tracking_grid_ptr = device_tracking_grid_ptrs[idx];
    // Free the device memory
    hipFree(device_tracking_grid_ptr);
    CudaCheckErrors("Failed to free device tracking grid");
  }
  // Free the device memory
  hipFree(device_filter_grid_ptr);
  CudaCheckErrors("Failed to free device filter grid");
}
}  // namespace cuda_helpers
}  // namespace pointcloud_voxelization
}  // namespace voxelized_geometry_tools
