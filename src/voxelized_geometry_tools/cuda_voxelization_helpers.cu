#include "hip/hip_runtime.h"
#include <voxelized_geometry_tools/cuda_voxelization_helpers.h>

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <cstdint>
#include <map>
#include <iostream>
#include <string>
#include <vector>

namespace voxelized_geometry_tools
{
namespace pointcloud_voxelization
{
namespace cuda_helpers
{
void CudaCheckErrors(const std::string& msg)
{
  const hipError_t last_error = hipGetLastError();
  if (last_error != hipSuccess)
  {
    const std::string error_string(hipGetErrorString(last_error));
    throw std::runtime_error("[" + msg + "] Cuda error [" + error_string + "]");
  }
}

__global__
void RaycastPoint(
    const float* const device_points_ptr, const int32_t num_points,
    const float* const device_pointcloud_origin_transform_ptr,
    const float* const device_tracking_grid_inverse_origin_transform_ptr,
    const float inverse_step_size, const float inverse_cell_size,
    const int32_t num_x_cells, const int32_t num_y_cells,
    const int32_t num_z_cells, const int32_t stride1, const int32_t stride2,
    int32_t* const device_tracking_grid_ptr)
{
  const int32_t point_index = blockIdx.x * blockDim.x + threadIdx.x;
  if (point_index < num_points)
  {
    const float ox = device_pointcloud_origin_transform_ptr[12];
    const float oy = device_pointcloud_origin_transform_ptr[13];
    const float oz = device_pointcloud_origin_transform_ptr[14];
    const float px = device_points_ptr[(point_index * 3) + 0];
    const float py = device_points_ptr[(point_index * 3) + 1];
    const float pz = device_points_ptr[(point_index * 3) + 2];
    const float wx = device_pointcloud_origin_transform_ptr[0] * px
                     + device_pointcloud_origin_transform_ptr[4] * py
                     + device_pointcloud_origin_transform_ptr[8] * pz
                     + device_pointcloud_origin_transform_ptr[12];
    const float wy = device_pointcloud_origin_transform_ptr[1] * px
                     + device_pointcloud_origin_transform_ptr[5] * py
                     + device_pointcloud_origin_transform_ptr[9] * pz
                     + device_pointcloud_origin_transform_ptr[13];
    const float wz = device_pointcloud_origin_transform_ptr[2] * px
                     + device_pointcloud_origin_transform_ptr[6] * py
                     + device_pointcloud_origin_transform_ptr[10] * pz
                     + device_pointcloud_origin_transform_ptr[14];
    const float rx = wx - ox;
    const float ry = wy - oy;
    const float rz = wz - oz;
    const float current_ray_length = sqrtf((rx * rx) + (ry * ry) + (rz * rz));
    const float num_steps =
        floor(current_ray_length * inverse_step_size);
    int32_t previous_x_cell = -1;
    int32_t previous_y_cell = -1;
    int32_t previous_z_cell = -1;
    for (float step = 0.0; step < num_steps; step += 1.0)
    {
      bool in_grid = false;
      const float elapsed_ratio = step / num_steps;
      const float cx = (rx * elapsed_ratio) + ox;
      const float cy = (ry * elapsed_ratio) + oy;
      const float cz = (rz * elapsed_ratio) + oz;
      const float gx =
          device_tracking_grid_inverse_origin_transform_ptr[0] * cx
          + device_tracking_grid_inverse_origin_transform_ptr[4] * cy
          + device_tracking_grid_inverse_origin_transform_ptr[8] * cz
          + device_tracking_grid_inverse_origin_transform_ptr[12];
      const float gy =
          device_tracking_grid_inverse_origin_transform_ptr[1] * cx
          + device_tracking_grid_inverse_origin_transform_ptr[5] * cy
          + device_tracking_grid_inverse_origin_transform_ptr[9] * cz
          + device_tracking_grid_inverse_origin_transform_ptr[13];
      const float gz =
          device_tracking_grid_inverse_origin_transform_ptr[2] * cx
          + device_tracking_grid_inverse_origin_transform_ptr[6] * cy
          + device_tracking_grid_inverse_origin_transform_ptr[10] * cz
          + device_tracking_grid_inverse_origin_transform_ptr[14];
      const int32_t x_cell = static_cast<int32_t>(gx * inverse_cell_size);
      const int32_t y_cell = static_cast<int32_t>(gy * inverse_cell_size);
      const int32_t z_cell = static_cast<int32_t>(gz * inverse_cell_size);
      if (x_cell != previous_x_cell || y_cell != previous_y_cell
          || z_cell != previous_z_cell)
      {
        if (x_cell >= 0 && x_cell < num_x_cells && y_cell >= 0
            && y_cell < num_y_cells && z_cell >= 0 && z_cell < num_z_cells)
        {
          in_grid = true;
          const int32_t cell_index =
              (x_cell * stride1) + (y_cell * stride2) + z_cell;
          // Increase free count
          atomicAdd(&(device_tracking_grid_ptr[cell_index * 2]), 1);
        }
        else if (in_grid)
        {
          // We've left the grid and there's no reason to keep going.
          break;
        }
      }
      previous_x_cell = x_cell;
      previous_y_cell = y_cell;
      previous_z_cell = z_cell;
    }
    // Set the point itself as filled
    const float gx =
        device_tracking_grid_inverse_origin_transform_ptr[0] * wx
        + device_tracking_grid_inverse_origin_transform_ptr[4] * wy
        + device_tracking_grid_inverse_origin_transform_ptr[8] * wz
        + device_tracking_grid_inverse_origin_transform_ptr[12];
    const float gy =
        device_tracking_grid_inverse_origin_transform_ptr[1] * wx
        + device_tracking_grid_inverse_origin_transform_ptr[5] * wy
        + device_tracking_grid_inverse_origin_transform_ptr[9] * wz
        + device_tracking_grid_inverse_origin_transform_ptr[13];
    const float gz =
        device_tracking_grid_inverse_origin_transform_ptr[2] * wx
        + device_tracking_grid_inverse_origin_transform_ptr[6] * wy
        + device_tracking_grid_inverse_origin_transform_ptr[10] * wz
        + device_tracking_grid_inverse_origin_transform_ptr[14];
    const int32_t x_cell = static_cast<int32_t>(gx * inverse_cell_size);
    const int32_t y_cell = static_cast<int32_t>(gy * inverse_cell_size);
    const int32_t z_cell = static_cast<int32_t>(gz * inverse_cell_size);
    if (x_cell >= 0 && x_cell < num_x_cells && y_cell >= 0
        && y_cell < num_y_cells && z_cell >= 0 && z_cell < num_z_cells)
    {
      const int32_t cell_index =
          (x_cell * stride1) + (y_cell * stride2) + z_cell;
      // Increase filled count
      atomicAdd(&(device_tracking_grid_ptr[(cell_index * 2) + 1]), 1);
    }
  }
}

__global__
void FilterGrids(
    const int64_t num_cells, const int32_t num_grids,
    const int32_t* const device_tracking_grids_ptr,
    float* const device_filter_grid_ptr, const float percent_seen_free,
    const int32_t outlier_points_threshold, const int32_t num_cameras_seen_free)
{
  const int32_t voxel_index = blockIdx.x * blockDim.x + threadIdx.x;
  if (voxel_index < num_cells)
  {
    const float current_occupancy = device_filter_grid_ptr[voxel_index * 2];
    // Filled cells stay filled, we don't work with them.
    // We only change cells that are unknown or empty.
    if (current_occupancy <= 0.5)
    {
      int32_t cameras_seen_filled = 0;
      int32_t cameras_seen_free = 0;
      for (int32_t idx = 0; idx < num_grids; idx++)
      {
        const int32_t* const device_tracking_grid_ptr =
            device_tracking_grids_ptr + (idx * num_cells * 2);
        const int32_t free_count = device_tracking_grid_ptr[voxel_index * 2];
        const int32_t filled_count =
            device_tracking_grid_ptr[(voxel_index * 2) + 1];
        const int32_t filtered_filled_count =
            (filled_count >= outlier_points_threshold) ? filled_count : 0;
        if (free_count > 0 && filtered_filled_count > 0)
        {
          const float current_percent_seen_free =
              static_cast<float>(free_count)
              / static_cast<float>(free_count + filtered_filled_count);
          if (current_percent_seen_free >= percent_seen_free)
          {
            cameras_seen_free += 1;
          }
          else
          {
            cameras_seen_filled += 1;
          }
        }
        else if (free_count > 0)
        {
          cameras_seen_free += 1;
        }
        else if (filtered_filled_count > 0)
        {
          cameras_seen_filled += 1;
        }
      }
      if (cameras_seen_filled > 0)
      {
        // If any camera saw something here, it is filled.
        device_filter_grid_ptr[voxel_index * 2] = 1.0;
      }
      else if (cameras_seen_free >= num_cameras_seen_free)
      {
        // Did enough cameras see this empty?
        device_filter_grid_ptr[voxel_index * 2] = 0.0;
      }
      else
      {
        // Otherwise, it is unknown.
        device_filter_grid_ptr[voxel_index * 2] = 0.5;
      }
    }
  }
}

int32_t RetrieveOptionOrDefault(
    const std::map<std::string, int32_t>& options, const std::string& option,
    const int32_t default_value)
{
  auto found_itr = options.find(option);
  if (found_itr != options.end())
  {
    const int32_t value = found_itr->second;
    std::cout << "Option [" << option << "] found with value [" << value << "]"
              << std::endl;
    return value;
  }
  else
  {
    std::cout << "Option [" << option << "] not found, default ["
              << default_value << "]" << std::endl;
    return default_value;
  }
}

class RealCudaVoxelizationHelperInterface
    : public CudaVoxelizationHelperInterface
{
public:
  explicit RealCudaVoxelizationHelperInterface(
      const std::map<std::string, int32_t>& options)
  {
    const int32_t cuda_device =
        RetrieveOptionOrDefault(options, "CUDA_DEVICE", 0);
    try
    {
      int32_t device_count = 0;
      hipGetDeviceCount(&device_count);
      CudaCheckErrors("Failed to get device count");
      if (cuda_device >= 0 && cuda_device < device_count)
      {
        cuda_device_num_ = cuda_device;
        SetCudaDevice();
      }
      else
      {
        std::cerr << "CUDA_DEVICE = " << cuda_device << " out of range for "
                  << device_count << " devices" << std::endl;
        cuda_device_num_ = -1;
      }
    }
    catch (const std::runtime_error& ex)
    {
      std::cerr << "Failed to load CUDA runtime and set device: "
                << ex.what() << std::endl;
      cuda_device_num_ = -1;
    }
  }

  ~RealCudaVoxelizationHelperInterface() override
  {
    CleanupAllocatedMemory();
  }

  bool IsAvailable() const override { return (cuda_device_num_ >= 0); }

  std::vector<int64_t> PrepareTrackingGrids(
      const int64_t num_cells, const int32_t num_grids) override
  {
    CleanupTrackingGridsMemory();
    const size_t tracking_grids_size =
        sizeof(int32_t) * 2 * num_cells * num_grids;
    hipMalloc(&device_tracking_grids_ptr_, tracking_grids_size);
    CudaCheckErrors("Failed to allocate device tracking grids");
    hipMemset(device_tracking_grids_ptr_, 0, tracking_grids_size);
    CudaCheckErrors("Failed to zero device tracking grids");
    std::vector<int64_t> tracking_grid_offsets(num_grids, 0);
    for (int32_t num_grid = 0; num_grid < num_grids; num_grid++)
    {
      tracking_grid_offsets.at(num_grid) = num_grid * num_cells * 2;
    }
    return tracking_grid_offsets;
  }

  void RaycastPoints(
      const std::vector<float>& raw_points,
      const float* const pointcloud_origin_transform,
      const float* const inverse_grid_origin_transform,
      const float inverse_step_size, const float inverse_cell_size,
      const int32_t num_x_cells, const int32_t num_y_cells,
      const int32_t num_z_cells,
      const int64_t tracking_grid_starting_offset) override
  {
    SetCudaDevice();
    const int32_t num_points = raw_points.size() / 3;
    // Copy the points
    const size_t points_size = sizeof(float) * raw_points.size();
    float* device_points_ptr = nullptr;
    hipMalloc(&device_points_ptr, points_size);
    CudaCheckErrors("Failed to allocate device points");
    hipMemcpy(device_points_ptr, raw_points.data(), points_size,
               hipMemcpyHostToDevice);
    CudaCheckErrors("Failed to memcpy the points to the device");
    // Copy pointcloud origin transform
    const size_t transform_size = sizeof(float) * 16;
    float* device_pointcloud_origin_transform_ptr = nullptr;
    hipMalloc(&device_pointcloud_origin_transform_ptr, transform_size);
    CudaCheckErrors("Failed to allocate device pointcloud origin transform");
    hipMemcpy(
        device_pointcloud_origin_transform_ptr, pointcloud_origin_transform,
        transform_size, hipMemcpyHostToDevice);
    CudaCheckErrors("Failed to memcpy the pointcloud origin transform");
    // Copy grid inverse origin transform
    float* device_tracking_grid_inverse_origin_transform_ptr = nullptr;
    hipMalloc(
        &device_tracking_grid_inverse_origin_transform_ptr, transform_size);
    CudaCheckErrors("Failed to allocate device grid inverse origin transform");
    hipMemcpy(
        device_tracking_grid_inverse_origin_transform_ptr,
        inverse_grid_origin_transform, transform_size, hipMemcpyHostToDevice);
    CudaCheckErrors("Failed to memcpy the grid inverse origin transform");
    // Prepare for raycasting
    const int32_t stride1 = num_y_cells * num_z_cells;
    const int32_t stride2 = num_z_cells;
    // Call the CUDA kernel
    const int32_t num_threads = 256;
    const int32_t num_blocks = (num_points + (num_threads - 1)) / num_threads;
    int32_t* const device_tracking_grid_ptr =
        device_tracking_grids_ptr_ + tracking_grid_starting_offset;
    RaycastPoint<<<num_blocks, num_threads>>>(
        device_points_ptr, num_points, device_pointcloud_origin_transform_ptr,
        device_tracking_grid_inverse_origin_transform_ptr,
        inverse_step_size, inverse_cell_size, num_x_cells, num_y_cells,
        num_z_cells, stride1, stride2, device_tracking_grid_ptr);
    // Free the device memory
    hipFree(device_points_ptr);
    CudaCheckErrors("Failed to free device points");
    hipFree(device_pointcloud_origin_transform_ptr);
    CudaCheckErrors("Failed to free device pointcloud origin transform");
    hipFree(device_tracking_grid_inverse_origin_transform_ptr);
    CudaCheckErrors(
        "Failed to free device tracking grid inverse origin tranform");
  }

  void PrepareFilterGrid(
       const int64_t num_cells, const void* host_data_ptr) override
  {
    CleanupFilterGridMemory();
    const size_t filter_grid_size = sizeof(float) * num_cells * 2;
    hipMalloc(&device_filter_grid_ptr_, filter_grid_size);
    CudaCheckErrors("Failed to allocate device filter grid");
    hipMemcpy(device_filter_grid_ptr_, host_data_ptr, filter_grid_size,
               hipMemcpyHostToDevice);
    CudaCheckErrors("Failed to memcpy the static environment to the device");
  }

  void FilterTrackingGrids(
       const int64_t num_cells, const int32_t num_grids,
       const float percent_seen_free, const int32_t outlier_points_threshold,
       const int32_t num_cameras_seen_free) override
  {
    // Call the CUDA kernel
    const int32_t num_threads = 256;
    const int32_t num_blocks = (num_cells + (num_threads - 1)) / num_threads;
    FilterGrids<<<num_blocks, num_threads>>>(
        num_cells, num_grids, device_tracking_grids_ptr_,
        device_filter_grid_ptr_, percent_seen_free, outlier_points_threshold,
        num_cameras_seen_free);
  }

  void RetrieveTrackingGrid(
      const int64_t num_cells, const int64_t tracking_grid_starting_index,
      void* host_data_ptr) override
  {
    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
    const size_t tracking_grid_size = sizeof(int32_t) * num_cells * 2;
    hipMemcpy(host_data_ptr,
               device_tracking_grids_ptr_ + tracking_grid_starting_index,
               tracking_grid_size, hipMemcpyDeviceToHost);
    CudaCheckErrors("Failed to memcpy the tracking grid back to the host");
  }

  void RetrieveFilteredGrid(
      const int64_t num_cells, void* host_data_ptr) override
  {
    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
    const size_t filter_grid_size = sizeof(float) * num_cells * 2;
    hipMemcpy(host_data_ptr, device_filter_grid_ptr_, filter_grid_size,
               hipMemcpyDeviceToHost);
    CudaCheckErrors("Failed to memcpy the filter grid back to the host");
  }

  void CleanupAllocatedMemory() override
  {
    CleanupTrackingGridsMemory();
    CleanupFilterGridMemory();
  }

  void SetCudaDevice()
  {
    hipSetDevice(cuda_device_num_);
    CudaCheckErrors("Failed to set device");
  }

private:
  void CleanupTrackingGridsMemory()
  {
    if (device_tracking_grids_ptr_ != nullptr)
    {
      hipFree(device_tracking_grids_ptr_);
      CudaCheckErrors("Failed to free device tracking grids");
      device_tracking_grids_ptr_ = nullptr;
    }
  }

  void CleanupFilterGridMemory()
  {
    if (device_filter_grid_ptr_ != nullptr)
    {
      hipFree(device_filter_grid_ptr_);
      CudaCheckErrors("Failed to free device filter grid");
      device_filter_grid_ptr_ = nullptr;
    }
  }

  int32_t cuda_device_num_ = -1;
  int32_t* device_tracking_grids_ptr_ = nullptr;
  float* device_filter_grid_ptr_ = nullptr;
};

CudaVoxelizationHelperInterface* MakeHelperInterface(
    const std::map<std::string, int32_t>& options)
{
  return new RealCudaVoxelizationHelperInterface(options);
}
}  // namespace cuda_helpers
}  // namespace pointcloud_voxelization
}  // namespace voxelized_geometry_tools
